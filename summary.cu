#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>

typedef struct group {
    // array of pointer of points
    int *p;

    // how many points in the group
    unsigned int length;
}Group;

typedef struct cluster {
    // array of pointer of groups
    Group** g;
    int num_point;
}Cluster;
int N;
int blockNum;

/*
 funct: create group and allocate the memory it needs
 parms:
 id: the id of group

 return:
 a pointer of Group
 */
Group* Create_Group(unsigned int id){
    Group* g = (Group*) malloc(sizeof(Group));
    g->p = (int*) malloc(1 * sizeof(int));
    g->length = 1;
    g->p[0] = id;

    return g;
}


/*
 funct: create cluster and allocate the memory it needs
 parms:
 n: the amount of points

 return:
 a pointer of Cluster
 */
Cluster* Create_Cluster(unsigned int n){
    Cluster* c = (Cluster*) malloc(sizeof(Cluster));
    c->g = (Group**) malloc(n * sizeof(Group*));

    c->num_point = n;

    int i;
    for(i = 0; i < n; ++i){
        c->g[i] = Create_Group(i);
    }

    return c;
}


/*
 funct: merge two groups
 parms:
 self: the group that absorb the other group
 other: the group that is going to be absorbed
 return:
 none
 */
void Merge_Group(Group* self, Group* other){
    unsigned int insert_position = self->length;
    self->length += other->length;
    self->p = (int*) realloc(self->p, self->length * sizeof(int));

    int i;
    for(i = 0; i < other->length; ++i){
        self->p[insert_position + i] = other->p[i];
    }

    free(other->p);
    free(other);
}

/*
 funct: merge two group in the cluster given by two group id
 parms:
 c: the cluster
 g1: group id of the group that will absorb the other group
 g2: group id of the group that will be absorbed
 return:
 none
 */
void Merge(Cluster* c, unsigned int g1, unsigned int g2){
    Merge_Group(c->g[g1], c->g[g2]);
    c->g[g2] = NULL;
}

// CUDA
#define BLOCK_SIZE 1024

__constant__ int Num;

float *adj, *hadj;
int length;
int *gids, *rgids, *hgids;

int *gab, *hgab;


__device__ int index(const int i, const int j) {return (2*Num-i-1)*i/2+j-i-1;}
int hindex(const int i, const int j) {return (2*N-i-1)*i/2+j-i-1;}

void ShowAdj(const float* adj) {
    printf("============Adj============\n");
    for (int i = 0; i < N-1; i++) {
        for (int j = i+1; j < N; j++) {
            printf("%.4lf ", hadj[hindex(i, j)]);
        }
        printf("\n");
    }
    printf("===========================\n");
}

void ShowCluster(const Cluster* c, const int* gids) {
    printf("==========Cluster==========\n");
    printf("Groups=%d\n", length);
    for (int g = 0; g < length; g++) {
        int gid = hgids[g];
        printf("[");
        for (int i = 0; i < c->g[gid]->length; i++) {
            if (i) printf(" ");
            printf("%d", c->g[gid]->p[i]);
        }
        printf("]\n");
    }
    printf("===========================\n");
}

void Clear(Cluster* c){
    int i;
    for(i = 0; i < c->num_point; ++i){
        if(c->g[i] == NULL){
            continue;
        }
        else{
            free(c->g[i]->p);
            free(c->g[i]);
        }
    }

    hipFree(gab);
    hipFree(adj);
    hipFree(gids);
    free(rgids);
    free(hgids);
    free(hgab);
    free(hadj);
}

void CheckError(const char name[]) {
    printf("%s\n", name);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));
}

double randn() {
    int sign = (rand()%10)>5?1:-1;
    return rand()%10000000/(double)10000000.*sign;
}

double matmul(double input1[], double input2[], int dim) {
    double dis = 0;
    for (int i=0; i<dim; i++) {
        dis += input1[i] * input2[i];
    }
    return dis;
}

__global__ void Update(float *adj, int *gids, int *gab, int length) {
    int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    if (idx >= length) return;
    int gid = gids[idx], ga = gab[0], gb = gab[1];
    if (gid < ga) {
        // printf("Update adj(%d, %d)\n", gid, ga);
        adj[index(gid, ga)] = max(adj[index(gid, ga)], adj[index(gid, gb)]);
    }
    else if (gid > ga && gid != gb) {
        // printf("Update adj(%d, %d)\n", gid, ga);
        int ra = min(gb, gid), rb = max(gb, gid);
        adj[index(ga, gid)] = max(adj[index(ga, gid)], adj[index(ra, rb)]);
    }
}

void Clusting() {
    double mind = 9999;
    for (int i = 0; i < length-1; i++) {
        for (int j = i+1; j < length; j++) {
            int gi = hgids[i], gj = hgids[j];
            double gd = hadj[hindex(gi, gj)];
            if (gd < mind) {
                mind = gd;
                hgab[0] = gi;
                hgab[1] = gj;
            }
        }
    }

    hipMemcpy(gab, hgab, 2*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gids, hgids, N*sizeof(int), hipMemcpyHostToDevice);
    // printf("Merge %d %d\n", hgab[0], hgab[1]);


    int BN = length/BLOCK_SIZE;
    if (length % BLOCK_SIZE) BN += 1;
    dim3 blockNum(BN);
    Update<<<blockNum, BLOCK_SIZE>>>(adj, gids, gab, length);
    // CheckError("update");
    hipDeviceSynchronize();
    // printf("Check U\n");
    hipMemcpy(hadj, adj, N*(N-1)/2*sizeof(float), hipMemcpyDeviceToHost);

    hgids[rgids[hgab[1]]] = hgids[length-1];
    rgids[hgids[length-1]] = rgids[hgab[1]];
    length -= 1;
}

int main(int argc, char *argv[]){
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
    N = atoi(argv[1]);
    hipMemcpyToSymbol(HIP_SYMBOL(Num), &N, sizeof(int));

    /*
     adjacent list, which store the distance
     ex. adj[0][1] is the distance between point 0 and point 1
     */
    hipMalloc((void**) &adj, N*(N-1)/2*sizeof(float));
    hiprandGenerateUniform(gen, adj, N*(N-1)/2);
    hipMalloc((void**) &gids, N*sizeof(int));
    hipMalloc((void**) &gab, 2*sizeof(int));
    hadj = (float*) malloc(N*(N-1)/2*sizeof(float));
    hipMemcpy(hadj, adj, N*(N-1)/2*sizeof(float), hipMemcpyDeviceToHost);
    rgids = (int*) malloc(N*sizeof(int));
    hgids = (int*) malloc(N*sizeof(int));
    hgab = (int*) malloc(2*sizeof(int));
    length = N;

    for (int i=0; i<N; ++i) {
      hgids[i] = i;
      rgids[i] = i;
    }

    Cluster *cluster = Create_Cluster(N);
    // ShowAdj(adj);
    // ShowCluster(cluster, gids);

    int cluster_size=1;
    while (length > cluster_size) {
        Clusting();

        Merge(cluster, hgab[0], hgab[1]);
        // ShowAdj(adj);
        // ShowCluster(cluster, gids);
    }

    // ShowCluster(cluster, gids);
    hiprandDestroyGenerator(gen);
    Clear(cluster);
}
